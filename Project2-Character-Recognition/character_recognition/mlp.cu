#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "mlp.h"

#define DIM 101
#define LABELS 52

#define checkCUDAErrorWithLine(msg) checkCUDAError(msg, __LINE__)

#define blockSize 128

namespace CharacterRecognition {
    using Common::PerformanceTimer;
    PerformanceTimer& timer()
    {
        static PerformanceTimer timer;
        return timer;
    }

	float *dev_input;
	float *dev_hidden;
	float *dev_output;
	float *dev_w_kj;
	float *dev_w_ki;

	int inputDims = DIM * DIM;
	int hiddenDims = inputDims;
	int outputDims = LABELS;

	__global__ void backprop() {
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n_output) {
			return;
		}
	}
        
	void train() {


		-PRODUCT(C7 - R9, (1 / (1 + EXP(-R9))), (1 - 1 / (1 + EXP(-R9))), O5)
		-PRODUCT(J5, 1 / (1 + EXP(-O5)), 1 - 1 / (1 + EXP(-O5)), C7 - R9, 1 / (1 + EXP(-R9)), 1 - 1 / (1 + EXP(-R9)), P8)
	}

	__global__ void kernComputeLayer(float *inputLr, float *outputLr, int n_input, int n_output, float *weights) {
		int index = threadIdx.x + (blockIdx.x * blockDim.x);
		if (index >= n_output) {
			return;
		}

		// Weighted sum of previous layer inputs
		for (int i = 0; i < n_input; i++) {
			int weightIndex = i + index * n_input;
			outputLr[index] += inputLr[i] * weights[weightIndex];
		}

		// Activation function
		outputLr[index] = 1 / (1 + expf(-outputLr[index]));
	}

	void printArray(const float *array, int n) {
		printf("[");
		for (int i = 0; i < n; i++) {
			printf("%f, ", array[i]);
		}
		printf("]\n");
	}

	void evaluate(float *input) {
		float *results = new float[outputDims];

		dim3 fullBlocksPerGridInToHid((inputDims * hiddenDims + blockSize - 1) / blockSize);
		dim3 fullBlocksPerGridHidToOut((hiddenDims * outputDims + blockSize - 1) / blockSize);

		hipMemcpy(dev_input, input, sizeof(float) * inputDims, hipMemcpyHostToDevice);

		kernComputeLayer << <fullBlocksPerGridInToHid, blockSize >> > (dev_input, dev_hidden, inputDims, hiddenDims, dev_w_kj);
		kernComputeLayer << <fullBlocksPerGridHidToOut, blockSize >> > (dev_hidden, dev_output, hiddenDims, outputDims, dev_w_ki);

		hipMemcpy(results, dev_output, sizeof(float) * outputDims, hipMemcpyDeviceToHost);

		printArray(results, outputDims);
		delete[] results;
	}

	void init() {
		dim3 fullBlocksPerGridInToHid((inputDims * hiddenDims + blockSize - 1) / blockSize);
		dim3 fullBlocksPerGridHidToOut((hiddenDims * outputDims + blockSize - 1) / blockSize);

		hipMalloc((void**)&dev_input, inputDims * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_input failed!");

		hipMalloc((void**)&dev_hidden, hiddenDims * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_hidden failed!");

		hipMalloc((void**)&dev_output, outputDims * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_output failed!");

		hipMalloc((void**)&dev_w_kj, inputDims * hiddenDims * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_w_kj failed!");

		hipMalloc((void**)&dev_w_ki, hiddenDims * outputDims * sizeof(float));
		checkCUDAErrorWithLine("hipMalloc dev_w_ki failed!");

		float *weights1 = new float[inputDims * hiddenDims];
		for (int i = 0; i < inputDims * hiddenDims; i++) {
			float r = ((double)rand() / (RAND_MAX));
			weights1[i] = r * 2.0f - 1.0f;
		}
		float *weights2 = new float[hiddenDims * outputDims];
		for (int i = 0; i < hiddenDims * outputDims; i++) {
			float r = ((double)rand() / (RAND_MAX));
			weights2[i] = r * 2.0f - 1.0f;
		}
		hipMemcpy(dev_w_kj, weights1, sizeof(float) * inputDims * hiddenDims, hipMemcpyHostToDevice);
		hipMemcpy(dev_w_ki, weights2, sizeof(float) * hiddenDims * outputDims, hipMemcpyHostToDevice);

		delete[] weights1;
		delete[] weights2;
	}

	void end() {
		hipFree(dev_input);
		hipFree(dev_hidden);
		hipFree(dev_output);
		hipFree(dev_w_kj);
		hipFree(dev_w_ki);
	}
}
